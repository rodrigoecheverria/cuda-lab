#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/adjacent_difference.h>
#include <thrust/generate.h>
#include <thrust/unique.h>
#include <thrust/scan.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>

#define SITES            10
#define MAX_MEASUREMENT   8

unsigned int TotalRain ( thrust::device_vector<unsigned int>& M)  
{ return thrust::reduce(M.begin(), M.end()); }

unsigned int TotalDaysRainInSite ( thrust::device_vector<unsigned int>& S, 
                                   const unsigned int Site)
{ return thrust::count(S.begin(), S.end(), Site); }

unsigned int TotalSites ( thrust::device_vector<unsigned int>& S)
{ 
  thrust::pair<thrust::device_vector<unsigned int>::iterator, thrust::device_vector<unsigned int>::iterator> new_end;
  thrust::device_vector<unsigned int> G(S.size());
  thrust::device_vector<unsigned int> D(S.size());
  thrust::device_vector<unsigned int> K(S.size());
  thrust::sort(S.begin(), S.end());
  
  new_end = thrust::reduce_by_key(S.begin(), S.end(), G.begin(),K.begin(), D.begin() );

  return new_end.first - K.begin();
}


typedef thrust::device_vector<int>::iterator   SiteIt;
typedef thrust::device_vector<int>::iterator   MeasureIt;
typedef thrust::tuple<SiteIt, MeasureIt> IteratorTuple;
typedef thrust::zip_iterator<IteratorTuple> ZipIterator;

struct zero_if_not_site : thrust::unary_function<thrust::tuple<int,int>,thrust::tuple<int,int> >
{
    const unsigned int site;
    zero_if_not_site(unsigned int _site) : site(_site) {}
    
    __host__ __device__ thrust::tuple<int,int> operator()(const thrust::tuple<int,int> &x) const
    {
      return thrust::get<0>(x) == site ? x : thrust::make_tuple(thrust::get<0>(x),0);
    }
};
struct add_tuple_value : thrust::binary_function<thrust::tuple<int,int>,thrust::tuple<int,int>,int>
{
  int operator()(const thrust::tuple<int,int> x, const thrust::tuple<int,int> y)
  {
     return thrust::get<1>(x) + thrust::get<1>(y);
  }
  
};
unsigned int TotalRainIN ( thrust::device_vector<unsigned int>& S, 
                           thrust::device_vector<unsigned int>& M, 
                           const unsigned int St)
  {  
    //ZipIterator iter(thrust::make_tuple(S.begin(), M.begin()));
    //ZipIterator result = thrust::partition(iter, iter.end(), in_site(St)); //see
    /*return thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(S.begin(), M.begin())),
                                    thrust::make_zip_iterator(thrust::make_tuple(S.end(),   M.end  ())),
                                    zero_if_not_site(St),0,
                                    add_tuple_value()
                                    );*/
  return thrust::reduce(thrust::make_zip_iterator(thrust::make_tuple(S.begin(), M.begin())),
                        thrust::make_zip_iterator(thrust::make_tuple(S.end(),   M.end  ())),
                        (unsigned int) 0,
                        add_tuple_value()
                                    );
                                    //return 0;
  }

unsigned int TotalRainBetween ( thrust::device_vector<unsigned int>& D, 
                                thrust::device_vector<unsigned int>& M, 
                                const unsigned int Start, const unsigned int End)
  { return 0; }

unsigned int TotalDaysWithRain ( thrust::device_vector<unsigned int>& D) { return 0; }

unsigned int TotalDaysRainHigher( thrust::device_vector<unsigned int>& D, 
                                  thrust::device_vector<unsigned int>& M, 
                                  const unsigned int Min)
  { return 0;}//thrust::count_if() }


bool Option ( char o, thrust::device_vector<unsigned int>& Days, 
                      thrust::device_vector<unsigned int>& Sites, 
                      thrust::device_vector<unsigned int>& Measurements) 
{
  switch (o) {
    case '0': std::cout << "Total Rainfall is " << TotalRain( Measurements ) << std::endl; break;

    case '1': std::cout << "Total number of Days with any Rainfall in Site 3: " 
                 << TotalDaysRainInSite ( Sites, 3 ) << std::endl;   break;

    case '2': std::cout << "Total Sites with rain: " << TotalSites ( Sites ) << std::endl; break;

    case '3': std::cout << "Total Rainfall in Site 7 is " << TotalRainIN ( Sites, Measurements, 7 )
                 << std::endl; break;

    case '4': std::cout << "Total Rainfall between days 7 and 77 is " 
                 << TotalRainBetween ( Days, Measurements, 7, 77 ) << std::endl; break;

    case '5': std::cout << "Total number of Days with any rainfall: " 
                << TotalDaysWithRain ( Days ) << std::endl;  break;

    case '6': std::cout << "Number of Days where Rainfall exceeded 10 is " 
                << TotalDaysRainHigher ( Days, Measurements, 10 ) << std::endl; break;

    default:  return false;
  }
  return true;
}

struct rand_modulus {
    unsigned int N;
    rand_modulus(unsigned int _NN) : N(_NN) {}

    __host__ __device__
        unsigned int operator()() const { 
            return rand() % N; //N*SITES
        }
};

struct is_equal {
    __host__ __device__
        unsigned int operator() ( const unsigned int& d, const unsigned int& s )  { 
            return d==s? 1: 0;
        }
};

struct get_site {
    __host__ __device__
        unsigned int operator() ( const unsigned int& v )  { 
            return v % SITES;
        }
};

struct get_day {
    __host__ __device__
        unsigned int operator() ( const unsigned int& v )  { 
            return v / SITES;
        }
};


unsigned int rand_mes() {
  return  (unsigned int) pow( 2.0, ((double) (rand() % 100000)) /  (100000 / MAX_MEASUREMENT) );
}


int main (int argc, char **argv)
{
  unsigned int N=20;
  char o= '1';
  int Dup = -1;

  if (argc>1) {  o = argv[1][0];  }
  if (argc>2) {  N = atoi(argv[2]); }
  
  if (o == 'H' || o == 'h') {
    std::cout <<  "Arguments: (H|1|2|3|4|5|6) N " << std::endl;
    exit(0);
  }

  // use this host vector to generate random input data
  thrust::host_vector<unsigned int> HDay(N);
  thrust::host_vector<unsigned int> HMes(N);

  srand(0); // init random generation seed: same random numbers generated in each execution

  // Generate Information sorted by (increasing) day and site, and with no duplicates (day, site)
  thrust::generate ( HDay.begin(), HDay.end(), rand_modulus(N*SITES) );
  thrust::generate ( HMes.begin(), HMes.end(), rand_mes ); 

  // Create Device vectors and copy data from host vectors
  thrust::device_vector<unsigned int> Days        = HDay;
  thrust::device_vector<unsigned int> Measurements= HMes;
  thrust::device_vector<unsigned int> Sites(N);

  // Sort data and modify to avoid duplicates ( only works fine if SITES=10 )
  thrust::sort ( Days.begin(), Days.end() ); 
  do {
    Dup++;
    thrust::transform ( Days.begin(), Days.end()-1, Days.begin()+1, Sites.begin(), is_equal() );
    thrust::transform ( Days.begin()+1, Days.end(), Sites.begin(), Days.begin()+1, thrust::plus<unsigned int>() ); 
  } while (thrust::reduce ( Sites.begin(), Sites.end()-1 ) > 0);

  thrust::transform ( Days.begin(), Days.end(), Sites.begin(), get_site() );
  thrust::transform ( Days.begin(), Days.end(), Days.begin(), get_day() );


  if (Dup >0)
     std::cout << "Phases to extract duplicates during generation: " << Dup << std::endl << std::endl;

  if ( N<=20 ) { // for small cases: print contains of input vectors
    std::cout << "Days:         ";
    thrust::copy( Days.begin(), Days.end(), std::ostream_iterator<unsigned int>( std::cout, ", " ));
    std::cout << std::endl << "Sites:        "; 
    thrust::copy( Sites.begin(), Sites.end(), std::ostream_iterator<unsigned int>( std::cout, ", " ));
    std::cout << std::endl << "Measurements: "; 
    thrust::copy( HMes.begin(), HMes.end(), std::ostream_iterator<unsigned int>( std::cout, ", " ));
    std::cout << std::endl;
  } 

  // create device vectors and copy data from host vectors

  Option ( o, Days, Sites, Measurements);
 
  return 0;
}
