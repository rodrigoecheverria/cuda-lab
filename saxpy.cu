#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <iostream>
#include <iterator>
#include <algorithm>

void saxpy_slow(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    thrust::device_vector<float> temp(X.size());
    thrust::fill(temp.begin(), temp.end(), A);
    thrust::transform(X.begin(), X.end(), temp.begin(), temp.begin(), thrust::multiplies<float>());
    thrust::transform(temp.begin(), temp.end(), Y.begin(), Y.begin(), thrust::plus<float>());
}

struct saxpy_functor : public thrust::binary_function<float,float,float>
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
        float operator()(const float& x, const float& y) const { 
            return a * x + y;
        }
};

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}


extern "C"
__global__ void saxpy ( const float *X, float *Y, float A, int N) 
{
  int i= threadIdx.x+blockDim.x*blockIdx.x;
  int T= blockDim.x*gridDim.x;
  #pragma unroll 4
  for (; i<N; i+=T)
    Y[i] = A*X[i] + Y[i];
}

#ifndef BLK_SZ
#define BLK_SZ 512
#endif

#ifndef THR
#define THR (BLK_SZ*6)
#endif

void saxpy_cuda(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
  saxpy <<< (THR-1)/BLK_SZ + 1, BLK_SZ >>> ( thrust::raw_pointer_cast(&(X[0])), 
  					     thrust::raw_pointer_cast(&(Y[0])), A, X.size());
}

int main(int argc, char **argv)
{
  // Default Size of input vectors
  int N=1000000;

  // Modify size of input vectors using program argument
  if (argc>1) {  N = atoi(argv[1]); }

  thrust::host_vector<float> x_h(N);
  thrust::host_vector<float> y_h(N);
  thrust::host_vector<float> y_result(N);

  thrust::sequence(x_h.begin(), x_h.end(), 10.0f, 1.5f);
  thrust::fill    (y_h.begin(), y_h.end(), -2.0f);

  thrust::device_vector<float> x(x_h.begin(), x_h.end());
  thrust::device_vector<float> y(y_h.begin(), y_h.end());

  thrust::transform(x_h.begin(), x_h.end(), y_h.begin(), y_h.begin(), saxpy_functor(2.0f));

  saxpy_slow(2.0f, x, y); 
  y_result = y;
  float R = thrust::inner_product( y_h.begin(), y_h.end(), y_result.begin(), 0.0f,
                                   thrust::plus<float>(), thrust::minus<float>());
  std::cout << "Saxpy Slow. Sum of Differences is " << R << std::endl;

  thrust::transform(x_h.begin(), x_h.end(), y_h.begin(), y_h.begin(), saxpy_functor(2.0f));
  saxpy_fast(2.0f, x, y);
  y_result = y;
  R = thrust::inner_product( y_h.begin(), y_h.end(), y_result.begin(), 0.0f,
                             thrust::plus<float>(), thrust::minus<float>());
  std::cout << "Saxpy Fast. Sum of Differences is " << R << std::endl;

  thrust::transform(x_h.begin(), x_h.end(), y_h.begin(), y_h.begin(), saxpy_functor(2.0f));
  saxpy_cuda(2.0f, x, y);
  y_result = y;
  R = thrust::inner_product( y_h.begin(), y_h.end(), y_result.begin(), 0.0f,
                             thrust::plus<float>(), thrust::minus<float>());
  std::cout << "Saxpy CUDA. Sum of Differences is " << R << std::endl;

  return 0;
}

